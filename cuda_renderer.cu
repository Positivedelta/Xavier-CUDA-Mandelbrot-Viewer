#include "hip/hip_runtime.h"
//
// (c) Max van Daalen, August 2020
//

#include <iostream>
#include "cuda_renderer.hpp"

//
// nvcc -std=c++14 -O3 -c cuda_renderer.cu -o cuda_renderer.o
//

__global__ void doMandelbrot(uint8_t *pixelBuffer, const int32_t pixelBufferSpan, const hipDoubleComplex start, const hipDoubleComplex step, const int32_t maxIterations)
{
    const int32_t x = threadIdx.x + blockIdx.x * blockDim.x;
    const int32_t y = threadIdx.y + blockIdx.y * blockDim.y;
    const hipDoubleComplex c = make_hipDoubleComplex(hipCreal(start) + (x * hipCreal(step)), hipCimag(start) + (y * hipCimag(step)));

    // notes 1. using fma() as it rounds coping with NaN and +/-  infinities, (zReal^2 + zImag^2) doesn't and eventually fails
    //       2. the GPU's fma(x, y, z) method efficiently computes (x * y) + z
    //       3. for speed, use (fabs(zReal) + fabs(zImag)) as an appromimation to norm(), however this does generate some minor artefacts
    //
    hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);
    int32_t iterations = maxIterations;
    while ((iterations > 0) && (fma(hipCreal(z), hipCreal(z), (hipCimag(z) * hipCimag(z))) < 4.0))
    {
        z = hipCadd(hipCmul(z, z), c);
        iterations--;
    }

    // using a smooth bernstein polynomial to generate the RGB components
    //
    const double t = (double)iterations / (double)maxIterations;
    const uint8_t r = (uint8_t)(9.0 * (1.0 - t) * t * t * t * 255.0);
    const uint8_t g = (uint8_t)(15.0 * (1.0 - t) * (1.0 - t) * t * t * 255.0);
    const uint8_t b = (uint8_t)(8.5 * (1.0 - t) * (1.0 - t) * (1.0 - t) * t * 255.0);

    // note, no need to update the alpha channel, it's expected to be pre-initialised outside of the kernel
    //
    const int32_t pixelAddr = (x << 2) + (y * pixelBufferSpan);
    pixelBuffer[pixelAddr] = r;
    pixelBuffer[pixelAddr + 1] = g;
    pixelBuffer[pixelAddr + 2] = b;
}

// note, the Xavier has 512 cores, hence using a block dimension of (32, 16)
//
CudaRenderer::CudaRenderer(const int32_t width, const int32_t height, const int32_t pixelBufferSpan):
    width(width), height(height), pixelBufferSpan(pixelBufferSpan), dimGrid(dim3(width / 32, height / 16)), dimBlock(dim3(32, 16)) {
        std::cout << "CUDA Kernel grid dimensions: (" << dimGrid.x << ", " << dimGrid.y << "), block: (" << dimBlock.x << ", " << dimBlock.y << ")\n";

        // used to time the doMandelbrot() kernel
        //
        hipEventCreate(&timerStart);
        hipEventCreate(&timerStop);

        const int32_t pixelBufferSize = pixelBufferSpan * height;
        std::cout << "Attempting to malloc " << pixelBufferSize << " bytes\n";

        hipError_t status = hipMallocManaged(&pixelBuffer, pixelBufferSize);
        switch (status)
        {
            case hipSuccess:
                std::cout << "Managed malloc successful\n";
                break;

            case hipErrorOutOfMemory:
                std::cout << "Unable to allocate enough memory\nProgram exiting!\n";
                doExit(status);
                break;

            case hipErrorNotSupported:
                std::cout << "Operation is not supported on the current device\nProgram exiting!\n";
                doExit(status);
                break;

            case hipErrorInvalidValue:
                std::cout << "One or more of the parameters passed to the API call is not within an acceptable range\nProgram exiting!\n";
                doExit(status);
                break;

            default:
                std::cout << "Managed malloc error: " << status << "\nProgram exiting!\n";
                doExit(status);
                break;
        }
}

CudaRenderer::~CudaRenderer()
{
    hipEventDestroy(timerStart);
    hipEventDestroy(timerStop);
    hipFree(pixelBuffer);
}

uint8_t* CudaRenderer::getPixelBuffer()
{
    return pixelBuffer;
}

void CudaRenderer::paintMandelbrot(const std::complex<double> start, const std::complex<double> end, const int32_t maxIterations)
{
    std::cout << "CUDA render starting...";
    const std::complex<double> range = end - start;
    const std::complex<double> step = std::complex<double>(range.real() / (double)width, range.imag() / (double)height);

    const hipDoubleComplex cuStart = make_hipDoubleComplex(start.real(), start.imag());
    const hipDoubleComplex cuStep = make_hipDoubleComplex(step.real(), step.imag());
    hipEventRecord(timerStart, 0);
    doMandelbrot<<<dimGrid, dimBlock>>>(pixelBuffer, pixelBufferSpan, cuStart, cuStep, maxIterations);
    hipEventRecord(timerStop, 0);
    const int32_t status = hipDeviceSynchronize();
    if (status == hipSuccess)
    {
        hipEventSynchronize(timerStop);

        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, timerStart, timerStop);
        std::cout << " completed in "<< elapsedTime << " ms\n\n";
    }
    else
    {
        std::cout << "\nCUDA device synchronisation failure: " << status << "\nProgram exiting!\n";
        doExit(status);
    }
}

void CudaRenderer::doExit(int32_t status)
{
    hipEventDestroy(timerStart);
    hipEventDestroy(timerStop);
    hipFree(pixelBuffer);

    // FIXME! not sure about the reset, it would probably mess up any other running applications that were using the GPU
    //
    hipDeviceReset();
    exit(status);
}
